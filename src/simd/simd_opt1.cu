#include "hip/hip_runtime.h"
// Copyright 2023. All Rights Reserved.
// Author: Bruce-Lee-LY
// Date: 21:02:28 on Tue, Feb 28, 2023
//
// Description: mma base hgemm

#include "common.h"

#define THREADS_PER_BLOCK 32 // WARP_SIZE * WARPS_PER_BLOCK
#define SMEM_SIZE 32

#define A(i,j) A[(j) + (i)*lda] // row major
#define B(i,j) B[(i) + (j)*ldb] // col major
#define C(i,j) C[(j) + (i)*ldc] // row major

#define sharedA(i,j) *(sharedA+(j)+(i)*SMEM_SIZE)
#define sharedB(i,j) *(sharedB+(j)+(i)*SMEM_SIZE)

__global__ void simdOpt1Kernel(const half *__restrict__ A, 
                              const half *__restrict__ B, 
                              half *__restrict__ C, 
                              size_t M, size_t N, size_t K) {

    // actually define two 32x32 shared mem for tile A and tile B
    extern __shared__ half sharedMem[];
    half* sharedA = sharedMem;
    half* sharedB = sharedMem + SMEM_SIZE*SMEM_SIZE;

    int lda=K, ldb=K, ldc=N;
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;

    A = &A((by<<5), 0);
    B = &B(0, (bx<<5));
    C = &C((by<<5), (bx<<5));

    //half tmp = 0.;
    float tmp = 0.;
    #pragma unroll
    for(int k=0; k<K; k+=SMEM_SIZE) {
        sharedA(ty, tx) = A(ty, tx);
        sharedB(ty, tx) = B(ty, tx);

        A += SMEM_SIZE;
        B += SMEM_SIZE;
        __syncthreads(); // wait for all threads in one block

        for(size_t i=0; i<SMEM_SIZE; ++i) {
            tmp += __half2float(sharedA(ty, i))*__half2float(sharedB(i, tx));
        }
        __syncthreads(); // wait for all threads in one block
    }


    //C(ty,tx) = tmp;
    C(ty,tx) = __float2half(tmp);
}

size_t initSimdOpt1() {
    int dev_id = 0;
    HGEMM_CHECK_CUDART_ERROR(hipGetDevice(&dev_id));

    hipDeviceProp_t dev_prop;
    HGEMM_CHECK_CUDART_ERROR(hipGetDeviceProperties(&dev_prop, dev_id));

    size_t smem_max_size = 2*SMEM_SIZE*SMEM_SIZE*sizeof(half);
    HLOG("smem_max_size: %.0f KBytes (%zu Bytes)", static_cast<double>(smem_max_size) / 1024, smem_max_size);

    HGEMM_CHECK_GT(dev_prop.sharedMemPerMultiprocessor, smem_max_size);
    HGEMM_CHECK_CUDART_ERROR(
        hipFuncSetAttribute(reinterpret_cast<const void*>(simdOpt1Kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_max_size));

    return smem_max_size;
}

void simdOpt1(half *A, half *B, half *C, size_t M, size_t N, size_t K) {
    static size_t smem_max_size = initSimdOpt1();

    dim3 block(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 grid(div_ceil(M, THREADS_PER_BLOCK), div_ceil(N, THREADS_PER_BLOCK));

    simdOpt1Kernel<<<grid, block, smem_max_size>>>(A, B, C, M, N, K);
}

