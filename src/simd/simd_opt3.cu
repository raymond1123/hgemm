#include "hip/hip_runtime.h"
// Copyright 2023. All Rights Reserved.
// Author: Bruce-Lee-LY
// Date: 21:02:28 on Tue, Feb 28, 2023
//
// Description: mma base hgemm

#include "common.h"

#define THREADS_PER_BLOCK 32 // WARP_SIZE * WARPS_PER_BLOCK
#define SMEM_SIZE 32
#define LOAD_NUM 8 // float4

#define A(i,j) A[(j)+(i)*lda] // row major
#define B(i,j) B[(i)+(j)*ldb] // col major
#define C(i,j) C[(j)+(i)*ldc] // row major

#define tileA(i,j) tileA[(j) + (i)*(lda/LOAD_NUM)] // row major
#define tileB(i,j) tileB[(i) + (j)*(ldb/LOAD_NUM)] // col major

#define sharedA(i,j) sharedA[(j)+(i)*SMEM_SIZE]
#define sharedB(i,j) sharedB[(j)+(i)*SMEM_SIZE]

__device__ void Opt3calc(const float4* loadA, 
                     const float4* loadB,
                     float* tmp) {

    half2 elem_a1 = *((half2*)loadA + 0);
    half2 elem_b1 = *((half2*)loadB + 0);
    half2 elem_a2 = *((half2*)loadA + 1);
    half2 elem_b2 = *((half2*)loadB + 1);
    half2 elem_a3 = *((half2*)loadA + 2);
    half2 elem_b3 = *((half2*)loadB + 2);
    half2 elem_a4 = *((half2*)loadA + 3);
    half2 elem_b4 = *((half2*)loadB + 3);

    half2 tmp1 = __hmul2(elem_a1, elem_b1);
    half2 tmp2 = __hmul2(elem_a2, elem_b2);
    half2 tmp3 = __hmul2(elem_a3, elem_b3);
    half2 tmp4 = __hmul2(elem_a4, elem_b4);

    half2 add_tmp1 = __hadd2(tmp1, tmp2);
    half2 add_tmp2 = __hadd2(tmp3, tmp4);
    half2 final_tmp3 = __hadd2(add_tmp1, add_tmp2);

    *tmp += __half2float(final_tmp3.x)+__half2float(final_tmp3.y);
}

__global__ void simdOpt3Kernel(const half *__restrict__ A, 
                              const half *__restrict__ B, 
                              half *__restrict__ C, 
                              size_t M, size_t N, size_t K) {

    // actually define two 32x32 shared mem for tile A and tile B
    extern __shared__ float4 sharedMem[];
    float4* sharedA = sharedMem;
    float4* sharedB = sharedMem + SMEM_SIZE*SMEM_SIZE;

    int lda=K, ldb=K, ldc=N;
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;

    float4* tileA = (float4*)(&A((by<<5), 0));
    float4* tileB = (float4*)(&B(0, (bx<<5)));
    C = &C((by<<5), (bx<<5));

    float tmp = 0.0;
    // in both smem ld or st, there should be no bankconflict at all.
    size_t iters = K/(SMEM_SIZE*LOAD_NUM);
    #pragma unroll
    for(size_t k=0; k<iters; ++k) {
        sharedA(ty, tx) = tileA(ty, tx);
        sharedB(ty, tx) = tileB(ty, tx);

        tileA += SMEM_SIZE;
        tileB += SMEM_SIZE;
        __syncthreads(); // wait for all threads in one block

        #pragma unroll
        for(size_t i=0; i<SMEM_SIZE; ++i) {
            float4* loadA = &(sharedA(ty,i));
            float4* loadB = &(sharedB(i,tx));

            Opt3calc(loadA, loadB, &tmp);

        }
        __syncthreads(); // wait for all threads in one block
    }

    C(ty,tx) = __float2half(tmp);
}

size_t initSimdOpt3() {
    int dev_id = 0;
    HGEMM_CHECK_CUDART_ERROR(hipGetDevice(&dev_id));

    hipDeviceProp_t dev_prop;
    HGEMM_CHECK_CUDART_ERROR(hipGetDeviceProperties(&dev_prop, dev_id));

    size_t smem_max_size = 2*SMEM_SIZE*SMEM_SIZE*sizeof(float4);
    HLOG("smem_max_size: %.0f KBytes (%zu Bytes)", static_cast<double>(smem_max_size) / 1024, smem_max_size);

    HGEMM_CHECK_GT(dev_prop.sharedMemPerMultiprocessor, smem_max_size);
    HGEMM_CHECK_CUDART_ERROR(
        hipFuncSetAttribute(reinterpret_cast<const void*>(simdOpt3Kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_max_size));

    return smem_max_size;
}

void simdOpt3(half *A, half *B, half *C, size_t M, size_t N, size_t K) {
    static size_t smem_max_size = initSimdOpt3();

    dim3 block(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 grid(div_ceil(M, THREADS_PER_BLOCK), div_ceil(N, THREADS_PER_BLOCK));

    simdOpt3Kernel<<<grid, block, smem_max_size>>>(A, B, C, M, N, K);
}

