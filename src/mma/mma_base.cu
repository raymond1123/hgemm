#include "hip/hip_runtime.h"
// Author: Raymond

#include "common.h"

__global__ void mmaBaseKernel(const half *__restrict__ A, 
                              const half *__restrict__ B, 
                              half *__restrict__ C, 
                              size_t M, size_t N, size_t K) {
    // m16n8k16
    const size_t M_tiles = ceil(M, MMA_M); 
    const size_t N_tiles = ceil(N, MMA_N);
    const size_t K_tiles = ceil(K, MMA_K);

    extern __shared__ half smem[][AB_SMEM_STRIDE];
    half* smemC = &smem[0][0];
    half* smemA = &smem[0][0];
    half* smemB = &smem[BLOCK_ROWS][0];

    // threadIdx.x = 0,1,...,255
    const size_t warp_id = threadIdx.x>>5;  // warp_id=0,1,...,7
    const size_t lane_id = threadIdx.x&31;

    // RC[4][8][2];
    uint32_t RC[WARP_COL_TILES][WARP_ROW_TILES][2];
    memset(RC, 0, sizeof(RC));

    /* step 1: block swizzle */
    size_t block_tile_i;
    size_t block_tile_j;
    swizzle(&block_tile_i, &block_tile_j);
    if (block_tile_i >= M_tiles || block_tile_j >= N_tiles) return;

    // address of the specific row loading into SRAM
    const half *A_warp_ptr = &A(block_tile_i, warp_id); 
    const half *B_warp_ptr = &B(block_tile_j, warp_id);

#pragma unroll
    for (size_t tile_k = 0; tile_k < K_tiles; tile_k += CHUNK_K) {

        /* step 2: load Matrix A & B from HBM to SRAM */
        ldgstsA_base(warp_id, lane_id, A_warp_ptr, tile_k, K, smemA);
        ldgstsB_base(warp_id, lane_id, B_warp_ptr, tile_k, K, smemB);

        __syncthreads();

        uint32_t RA[WARP_COL_TILES][4]; // RA[4][4]
        uint32_t RB[WARP_ROW_TILES][2]; // RB[8][2]

        #pragma unroll
        for (size_t k_step = 0; k_step < CHUNK_K; ++k_step) {

            /* step 3: load Matrix A & B from SRAM to Register */
            #pragma unroll
            for (size_t i = 0; i < WARP_COL_TILES; ++i) 
                ldsA_base(i, k_step, warp_id, lane_id, smemA, RA);

            #pragma unroll
            for (size_t j = 0; j < WARP_ROW_TILES; ++j)
                ldsB_base(j, k_step, warp_id, lane_id, smemB, RB);

            /* step 4: calc mma C=A@B */
            #pragma unroll
            for (size_t i = 0; i < WARP_COL_TILES; ++i) {
                #pragma unroll
                for (size_t j = 0; j < WARP_ROW_TILES; ++j)
                    hm16n8k16(i, j, RA, RB, RC);
            }
        }

        __syncthreads();
    }

    /* 
        step 5: load result from Register to SRAM

        every block has 8 warps, split into 4 rows and 2 cols 
        every warp holds 64x64 half elements
        (warp_id>>1)<<13 represents how many elements jumped over
        left shift 13 stands for 64*128
    */
    half *smem_warp_tile_row_ptr = smemC + ((warp_id>>1)<<13);
    #pragma unroll
    for (size_t i = 0; i < WARP_COL_TILES; ++i) { // i=0,1,2,3
        #pragma unroll
        for (size_t j = 0; j < WARP_ROW_TILES; ++j)  // j=0,1,...,7
            stsC_base(i, j, warp_id, lane_id, smem_warp_tile_row_ptr, RC);
    }

    __syncthreads();

    const size_t gmem_idx = (block_tile_i + warp_id * 2) * MMA_M * N + block_tile_j * MMA_N;
    const half *src_gmem_warp_stream_ptr = &C[gmem_idx];
    const half *smem_warp_stream_ptr = smemC + warp_id*2*MMA_M*C_SMEM_STRIDE;

    /* step 6: load result from SRAM to HBM */
    #pragma unroll
    for (size_t i = 0; i < MMA_M; ++i)
        ldsC_base(i, N, lane_id, src_gmem_warp_stream_ptr,smem_warp_stream_ptr);
}

size_t initMmaBase() {
    int dev_id = 0;
    HGEMM_CHECK_CUDART_ERROR(hipGetDevice(&dev_id));

    hipDeviceProp_t dev_prop;
    HGEMM_CHECK_CUDART_ERROR(hipGetDeviceProperties(&dev_prop, dev_id));

    size_t smem_max_size =
        std::max((BLOCK_ROWS + BLOCK_COLS) * AB_SMEM_STRIDE * sizeof(half), 
                  BLOCK_ROWS * C_SMEM_STRIDE * sizeof(half));
    HLOG("smem_max_size: %.0f KBytes (%zu Bytes)", static_cast<double>(smem_max_size) / 1024, smem_max_size);

    HGEMM_CHECK_GT(dev_prop.sharedMemPerMultiprocessor, smem_max_size);
    HGEMM_CHECK_CUDART_ERROR(
        hipFuncSetAttribute(reinterpret_cast<const void*>(mmaBaseKernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_max_size));

    return smem_max_size;
}

void mmaBase(half *A, half *B, half *C, size_t M, size_t N, size_t K) {
    static size_t smem_max_size = initMmaBase();

    dim3 block(THREADS_PER_BLOCK);
    dim3 grid(BLOCK_STRIDE, ceil(M, BLOCK_ROWS), ceil(N, BLOCK_COLS * BLOCK_STRIDE));

    mmaBaseKernel<<<grid, block, smem_max_size>>>(A, B, C, M, N, K);
}
